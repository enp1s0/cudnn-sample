
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <hip/hip_fp16.h>

constexpr std::size_t size = 1lu << 20;

int main() {
	float *dA;
	half  *dB;

	hipMalloc(&dA, sizeof(float) * size);
	hipMalloc(&dB, sizeof(half ) * size);

	cudnnHandle_t cudnn_handle;
	cudnnCreate(&cudnn_handle);

	cudnnTensorDescriptor_t a_tensor_desc;
	cudnnTensorDescriptor_t b_tensor_desc;
	cudnnCreateTensorDescriptor(&a_tensor_desc);
	cudnnCreateTensorDescriptor(&b_tensor_desc);

	cudnnSetTensor4dDescriptorEx(
			a_tensor_desc,
			CUDNN_DATA_FLOAT,
			1, size,
			1, 1,
			size, 1, 1, 1);
	cudnnSetTensor4dDescriptorEx(
			b_tensor_desc,
			CUDNN_DATA_HALF,
			1, size,
			1, 1,
			size, 1, 1, 1);

	const float alpha = 1.f, beta = 0.f;
	cudnnTransformTensor(
			cudnn_handle,
			&alpha,
			a_tensor_desc,
			dA, &beta,
			b_tensor_desc,
			dB
			);

	cudnnDestroyTensorDescriptor(a_tensor_desc);
	cudnnDestroyTensorDescriptor(b_tensor_desc);

	cudnnDestroy(cudnn_handle);

	hipFree(dA);
	hipFree(dB);
}
